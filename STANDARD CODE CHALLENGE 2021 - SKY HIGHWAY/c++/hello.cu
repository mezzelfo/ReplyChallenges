
#include <hip/hip_runtime.h>
#include <stdio.h>
//#define PESSIMISTIC_DIM (512)

struct Problem
{
    int H, W, N, M, R;
    int *latency_map;
    int *connection_speed_map;
    int *antennas_range;
    int *antennas_speed;
};

Problem read_file(const char *filename)
{
    FILE *fileptr = fopen(filename, "r");
    if (!fileptr)
    {
        printf("ERROR: File %s not found\n", filename);
        exit(-1);
    }

    Problem P;

    fscanf(fileptr, "%d %d %d %d %d", &(P.W), &(P.H), &(P.N), &(P.M), &(P.R));

    //Allocation
    P.latency_map = (int *)calloc(P.W * P.H, sizeof(int));
    P.connection_speed_map = (int *)calloc(P.W * P.H, sizeof(int));
    P.antennas_range = (int *)malloc(P.M * sizeof(int));
    P.antennas_speed = (int *)malloc(P.M * sizeof(int));

    if ((P.latency_map == NULL) || (P.connection_speed_map == NULL) || (P.antennas_range == NULL) || (P.antennas_speed == NULL))
    {
        printf("ERROR: Unable to malloc\n");
        exit(-2);
    }

    // Filling data structures
    for (int i = 0; i < P.N; i++)
    {
        int c, r, l, s;
        fscanf(fileptr, "%d %d %d %d\n", &c, &r, &l, &s);
        P.latency_map[P.W * r + c] = l;
        P.connection_speed_map[P.W * r + c] = s;
    }
    for (int i = 0; i < P.M; i++)
    {
        int r, s;
        fscanf(fileptr, "%d %d\n", &r, &s);
        P.antennas_range[i] = r;
        P.antennas_speed[i] = s;
    }
    fclose(fileptr);

    return P;
}

__global__ void optimisticEval(const Problem dev_P, const int ant_range, const int ant_speed, int *optimistic_score_map)
{
    const int r = blockIdx.x;
    const int c = threadIdx.x;

    //const int r = (blockIdx.x * blockDim.x) + threadIdx.x;
    //const int c = (blockIdx.y * blockDim.y) + threadIdx.y;

    int sum = 0;
    for (int rr = max(0, r - ant_range); rr <= min(dev_P.H - 1, r + ant_range); rr++)
    {
        const int dif = ant_range - abs(r - rr);
        for (int cc = max(0, c - dif); cc <= min(dev_P.W - 1, c + dif); cc++)
        {
            const int dist = abs(r - rr) + abs(c - cc);
            const int long contrib = dev_P.connection_speed_map[dev_P.W * rr + cc] * ant_speed - dev_P.latency_map[dev_P.W * rr + cc] * dist;
            if (contrib > 0)
            {
                sum += contrib;
            }
        }
    }
    optimistic_score_map[dev_P.W * r + c] = sum;
}

int cmpfunc(const void *a, const void *b)
{
    return (*(int *)a - *(int *)b);
}

int main(int argc, char const *argv[])
{
    printf("Inizio lettura file\n");
    Problem P = read_file(argv[1]);
    printf("File letto correttamente\n");

    // Copy only W H N M R
    Problem dev_P = P;
    dev_P.latency_map = NULL;
    dev_P.connection_speed_map = NULL;
    dev_P.antennas_range = NULL;
    dev_P.antennas_speed = NULL;

    // Output buffers
    int *optimistic_score_map, *dev_optimistic_score_map;
    //optimistic_score_map = (int *)malloc(P.W * P.H * sizeof(int));
    hipHostAlloc((void **)&optimistic_score_map, P.W * P.H * sizeof(int), hipHostMallocDefault);

    hipMalloc((void **)&(dev_P.latency_map), P.W * P.H * sizeof(int));
    hipMalloc((void **)&(dev_P.connection_speed_map), P.W * P.H * sizeof(int));
    hipMalloc((void **)&dev_optimistic_score_map, P.W * P.H * sizeof(int));
    hipMemcpy(dev_P.latency_map, P.latency_map, P.W * P.H * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.connection_speed_map, P.connection_speed_map, P.W * P.H * sizeof(int), hipMemcpyHostToDevice);

    //long int best_scores[PESSIMISTIC_DIM] = {0};
    //long int *cumulative_optimistic_score_map = (long int *)malloc(P.W * P.H * sizeof(long int));
    //int *antennas_contrib = (int*)malloc(P.M * sizeof(int));
    long int total = 0;
    for (int a = 0; a < P.M; a++)
    {
        printf("%d/%d\n", a, P.M);
        int ant_range = P.antennas_range[a];
        int ant_speed = P.antennas_speed[a];
        //optimisticEval<<<P.H, P.W>>>(dev_P, ant_range, ant_speed, dev_optimistic_score_map);
        dim3 threadsPerBlock(32, 32);
        dim3 numBlocks(P.W / threadsPerBlock.x, P.H / threadsPerBlock.y);
        optimisticEval<<<numBlocks, threadsPerBlock>>>(dev_P, ant_range, ant_speed, dev_optimistic_score_map);
        hipMemcpy(optimistic_score_map, dev_optimistic_score_map, P.W * P.H * sizeof(int), hipMemcpyDeviceToHost);
        int max_val = 0;
        for (size_t i = 0; i < P.H * P.W; i++)
        {
            if (optimistic_score_map[i] > max_val)
            {
                max_val = optimistic_score_map[i];
            }
        }
        total += max_val;
        //antennas_contrib[a] = max_val;
        /*qsort(optimistic_score_map, P.W * P.H, sizeof(int), cmpfunc);

        for (size_t i = 0; i < P.H * P.W; i++)
        {
            cumulative_optimistic_score_map[i] += optimistic_score_map[i];
        }*/

        /*long int best_contribs[PESSIMISTIC_DIM] = {0};
        for (int r = 0; r < P.H; r++)
        {
            for (int c = 0; c < P.W; c++)
            {
                int v = optimistic_score_map[P.W * r + c];
                if (v > best_contribs[0])
                {
                    int new_v = 1;
                    for (int i = 0; i < PESSIMISTIC_DIM; i++)
                    {
                        if (best_contribs[i] == v)
                        {
                            new_v = 0;
                            break;
                        }
                    }
                    if (new_v)
                    {
                        best_contribs[0] = v;
                        qsort(best_contribs, PESSIMISTIC_DIM, sizeof(long), cmpfunc);
                    }
                }
            }
        }
        for (int i = 0; i < PESSIMISTIC_DIM; i++)
        {
            best_scores[i] += best_contribs[i];
        }*/
    }
    /*for (int i = PESSIMISTIC_DIM - 1; i >= 0; i--)
    {
        //printf("%d-th optimistic_score: %ld\n", PESSIMISTIC_DIM-i, best_scores[i]);
        printf("%ld, ", best_scores[i]);
    }*/

    // FILE *fout = fopen("fout.csv", "w");

    // for (int a = 0; a < P.M; a++)
    // {
    //     fprintf(fout, "%d,", antennas_contrib[a]);
    // }

    /*for (size_t i = 0; i < P.H * P.W; i++)
    {
        fprintf(fout, "%ld,", cumulative_optimistic_score_map[i]);
    }
    free(cumulative_optimistic_score_map);*/

    //fclose(fout);

    printf("%ld\n", total);

    free(P.latency_map);
    free(P.connection_speed_map);
    free(P.antennas_range);
    free(P.antennas_speed);

    hipFree(dev_P.latency_map);
    hipFree(dev_P.connection_speed_map);

    hipHostFree(optimistic_score_map);
    free(optimistic_score_map);
    //cudaFree(dev_optimistic_score_map);

    return 0;
}