// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>

struct Antenna
{
    int range, speed, ID; //const?
    int x, y;
};

// int compareAntennas (const void * a, const void * b) {
//    return ( ((Antenna*)b)->range - ((Antenna*)a)->range); //Sort by decreasing range
// }

struct Building
{
    int x, y, latency, speed; //const?
    int antenna_connected_to;
    int best_score;
};

__global__ void solutionEval(Building *buildings, const Antenna *antennas, const int M)
{
    const int build_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const Building b = buildings[build_idx];
    long int best_score = 0;
    int antenna_connected_to = -1;
    for (size_t a_idx = 0; a_idx < M; a_idx++)
    {
        const Antenna a = antennas[a_idx];
        const int dist = abs(b.x - a.x) + abs(b.y - a.y);
        if (dist <= a.range)
        {
            const int contrib = b.speed * a.speed - b.latency * dist;
            if ((contrib > best_score) | (antenna_connected_to == -1))
            {
                best_score = contrib;
                antenna_connected_to = a_idx;
            }
        }
    }
    buildings[build_idx].best_score = best_score;
    buildings[build_idx].antenna_connected_to = antenna_connected_to;
}

__global__ void antennaPositionEval(const Antenna a, const int H, const int W, const Building *buildings, const int *dev_buildings_map, int *score_map)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    //printf("(%d,%d)\n",y,x);

    int score = 0;
    for (int yy = max(0, y - a.range); yy <= min(H - 1, y + a.range); yy++)
    {
        const int dif = a.range - abs(y - yy);
        for (int xx = max(0, x - dif); xx <= min(W - 1, x + dif); xx++)
        {
            const int build_idx = dev_buildings_map[W * yy + xx];
            if (build_idx != -1)
            {
                Building b = buildings[build_idx];
                const int dist = abs(y - yy) + abs(x - xx);
                const int contrib = b.speed * a.speed - b.latency * dist;
                if (contrib > b.best_score)
                {
                    score += (contrib - b.best_score);
                }
                else if (b.antenna_connected_to == -1)
                {
                    score += contrib;
                }
            }
        }
    }
    //printf("called host with (%d,%d): %d\n",r,c,score);
    score_map[W * y + x] = score;
}

int main(int argc, char const *argv[])
{
    /* 
    create random solution
    load data on GPU

    for iteration {
        select an antenna (randomly?)
        get best position for that antenna
        move that antenna & adjust solution
    }
     */

    Building *buildings, *dev_buildings;
    Antenna *antennas, *dev_antennas;

    int *buildings_map, *score_map;
    int *dev_buildings_map, *dev_score_map;

    int W, H, N, M, R;
    std::ifstream file(argv[1], std::ios::in);
    file >> W >> H >> N >> M >> R;

    hipHostAlloc((void **)&buildings, N * sizeof(Building), hipHostMallocDefault);
    hipHostAlloc((void **)&antennas, M * sizeof(Antenna), hipHostMallocDefault);
    hipMalloc((void **)&(dev_buildings), N * sizeof(Building));
    hipMalloc((void **)&(dev_antennas), M * sizeof(Antenna));

    buildings_map = (int *)malloc(H * W * sizeof(int));
    score_map = (int *)malloc(H * W * sizeof(int));
    memset(buildings_map, -1, H * W * sizeof(int));

    hipMalloc((void **)&(dev_buildings_map), H * W * sizeof(int));
    hipMalloc((void **)&(dev_score_map), H * W * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        int x, y, l, s;
        file >> x >> y >> l >> s;
        buildings[i].x = x;
        buildings[i].y = y;
        buildings[i].latency = l;
        buildings[i].speed = s;
        buildings[i].antenna_connected_to = -1;
        buildings[i].best_score = INT_MIN;
        buildings_map[W * y + x] = i;
    }
    srand(time(NULL));
    for (int i = 0; i < M; i++)
    {
        int r, s;
        file >> r >> s;
        antennas[i].ID = i;
        antennas[i].range = r;
        antennas[i].speed = s;
        antennas[i].x = rand() % W;
        antennas[i].y = rand() % H;
    }
    file.close();

    //qsort(antennas, M, sizeof(Antenna),compareAntennas);

    hipMemcpy(dev_buildings_map, buildings_map, H * W * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_buildings, buildings, N * sizeof(Building), hipMemcpyHostToDevice);
    hipMemcpy(dev_antennas, antennas, M * sizeof(Antenna), hipMemcpyHostToDevice);

    const dim3 threadsPerBlock(8, 8);
    const dim3 numBlocks(W / threadsPerBlock.x, H / threadsPerBlock.y);

    for (size_t iteration = 0; iteration < 10000; iteration++)
    {
        solutionEval<<<N / 1000, 1000>>>(dev_buildings, dev_antennas, M);

        if (iteration % 100 == 0)
        {
            hipMemcpy(buildings, dev_buildings, N * sizeof(Building), hipMemcpyDeviceToHost);
            long int score = 0;
            for (int b = 0; b < N; b++)
            {
                if (buildings[b].antenna_connected_to != -1)
                {
                    score += buildings[b].best_score;
                }
            }
            std::cout << (score * 1.0) / (2078043619.0) << std::endl; //B
            //std::cout << (score * 1.0) / (5247238794.0) << std::endl; //D
            //std::cout << (score * 1.0) / (8109310667.0) << std::endl; //F
        }

        //Find best position for antenna
        int a_idx = rand() % M;
        antennaPositionEval<<<numBlocks, threadsPerBlock>>>(antennas[a_idx], H, W, dev_buildings, dev_buildings_map, dev_score_map);
        hipMemcpy(score_map, dev_score_map, H * W * sizeof(int), hipMemcpyDeviceToHost);
        int pos_x = -1;
        int pos_y = -1;
        int best = 0;
        for (int x = 0; x < W; x++)
        {
            for (int y = 0; y < H; y++)
            {
                if ((score_map[W * y + x] > best) | (pos_x == -1))
                {
                    best = score_map[W * y + x];
                    pos_x = x;
                    pos_y = y;
                }
            }
        }

        //Move antenna
        antennas[a_idx].x = pos_x;
        antennas[a_idx].y = pos_y;
        hipMemcpy(dev_antennas, antennas, M * sizeof(Antenna), hipMemcpyHostToDevice);
    }

    // for (size_t a = 0; a < M; a++)
    // {
    //     std::cout << antennas[a].ID << ": (" << antennas[a].x << "," << antennas[a].y << ")\n";
    // }

    hipHostFree(buildings);
    hipHostFree(antennas);
    hipFree(dev_buildings);
    hipFree(dev_antennas);

    free(buildings_map);
    free(score_map);
    hipFree(dev_buildings_map);
    hipFree(dev_score_map);

    return 0;
}
