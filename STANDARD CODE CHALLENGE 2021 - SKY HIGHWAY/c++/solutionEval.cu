#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

struct Problem
{
    int H, W, N, M, R;
    int *buildings_c;
    int *buildings_r;
    int *buildings_latency;
    int *buildings_connection_speed;
    int *antennas_range;
    int *antennas_speed;
};


Problem read_file(const char *filename)
{
    FILE *fileptr = fopen(filename, "r");
    if (!fileptr)
    {
        printf("ERROR: File %s not found\n", filename);
        exit(-1);
    }

    Problem P;

    fscanf(fileptr, "%d %d %d %d %d", &(P.W), &(P.H), &(P.N), &(P.M), &(P.R));

    //Allocation
    P.buildings_c = (int *)calloc(P.N, sizeof(int));
    P.buildings_r = (int *)calloc(P.N, sizeof(int));
    P.buildings_latency = (int *)calloc(P.N, sizeof(int));
    P.buildings_connection_speed = (int *)calloc(P.N, sizeof(int));
    P.antennas_range = (int *)malloc(P.M * sizeof(int));
    P.antennas_speed = (int *)malloc(P.M * sizeof(int));

    if ((P.buildings_c == NULL) || (P.buildings_r == NULL) || (P.buildings_latency == NULL) || (P.buildings_connection_speed == NULL) || (P.antennas_range == NULL) || (P.antennas_speed == NULL))
    {
        printf("ERROR: Unable to malloc\n");
        exit(-2);
    }

    // Filling data structures
    for (int i = 0; i < P.N; i++)
    {
        int c, r, l, s;
        fscanf(fileptr, "%d %d %d %d\n", &c, &r, &l, &s);
        P.buildings_c[i] = c;
        P.buildings_r[i] = r;
        P.buildings_latency[i] = l;
        P.buildings_connection_speed[i] = s;
    }
    for (int i = 0; i < P.M; i++)
    {
        int r, s;
        fscanf(fileptr, "%d %d\n", &r, &s);
        P.antennas_range[i] = r;
        P.antennas_speed[i] = s;
    }
    fclose(fileptr);

    return P;
}

__global__ void solutionEval(const Problem dev_P, const int *antennas_positions, int *building_score, int *building_antenna)
{
    const int build_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int antenna_idx = -1;
    int max_contribution = 0;
    for (size_t a = 0; a < dev_P.M; a++)
    {
        //Naive implementation
        // const int dist = abs(dev_P.buildings_c[build_idx] - antennas_positions[2 * a]) + abs(dev_P.buildings_r[build_idx] - antennas_positions[2 * a + 1]);
        // if (dist <= dev_P.antennas_range[a])
        // {
        //     const int long contrib = dev_P.buildings_connection_speed[build_idx] * dev_P.antennas_speed[a] - dev_P.buildings_latency[build_idx] * dist;
        //     if (contrib > max_contribution)
        //     {
        //         max_contribution = contrib;
        //         antenna_idx = a;
        //     }
        // }
        //Suddivido in due il calcolo di dist per efficienza
        int dist = abs(dev_P.buildings_c[build_idx] - antennas_positions[2 * a]);
        if (dist <= dev_P.antennas_range[a])
        {
            dist += abs(dev_P.buildings_r[build_idx] - antennas_positions[2 * a + 1]);
            if (dist <= dev_P.antennas_range[a])
            {
                int contrib = dev_P.buildings_connection_speed[build_idx] * dev_P.antennas_speed[a] - dev_P.buildings_latency[build_idx] * dist;
                if (contrib > max_contribution)
                {
                    max_contribution = contrib;
                    antenna_idx = a;
                }
            }
        }
    }
    building_score[build_idx] = max_contribution;
    building_antenna[build_idx] = antenna_idx;
    //printf("Chiamata GPU per building %d\nMax contribution %d\n Best antenna %d\n\n", build_idx, max_contribution, antenna_idx);
}

int main(int argc, char const *argv[])
{
    printf("Inizio lettura file\n");
    Problem P = read_file(argv[1]);
    printf("File letto correttamente\n");

    // Copy only W H N M R
    Problem dev_P;
    dev_P.W = P.W;
    dev_P.H = P.H;
    dev_P.N = P.N;
    dev_P.M = P.M;
    dev_P.R = P.R;

    hipMalloc((void **)&(dev_P.buildings_c), P.N * sizeof(int));
    hipMalloc((void **)&(dev_P.buildings_r), P.N * sizeof(int));
    hipMalloc((void **)&(dev_P.buildings_latency), P.N * sizeof(int));
    hipMalloc((void **)&(dev_P.buildings_connection_speed), P.N * sizeof(int));
    hipMalloc((void **)&(dev_P.antennas_range), P.M * sizeof(int));
    hipMalloc((void **)&(dev_P.antennas_speed), P.M * sizeof(int));

    hipMemcpy(dev_P.buildings_c, P.buildings_c, P.N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.buildings_r, P.buildings_r, P.N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.buildings_latency, P.buildings_latency, P.N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.buildings_connection_speed, P.buildings_connection_speed, P.N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.antennas_range, P.antennas_range, P.M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_P.antennas_speed, P.antennas_speed, P.M * sizeof(int), hipMemcpyHostToDevice);

    // Output buffers
    int *antennas_positions, *building_score, *building_antenna;
    int *dev_antennas_positions, *dev_building_score, *dev_building_antenna;
    hipHostAlloc((void **)&antennas_positions, 2 * P.M * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_antennas_positions, 2 * P.M * sizeof(int));
    hipHostAlloc((void **)&building_score, P.N * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_building_score, P.N * sizeof(int));
    hipHostAlloc((void **)&building_antenna, P.N * sizeof(int), hipHostMallocDefault);
    hipMalloc((void **)&dev_building_antenna, P.N * sizeof(int));

    srand(time(NULL));

    long int best_score = 0;
    int *best_antennas_positions = (int *)malloc(2 * P.M * sizeof(int));

    for (size_t times = 0; times < 100; times++)
    {
        //Assign solution
        for (size_t a = 0; a < P.M; a++)
        {
            antennas_positions[2 * a] = rand() % P.W;
            antennas_positions[2 * a + 1] = rand() % P.H;
        }

        //Copy solution to GPU
        hipMemcpy(dev_antennas_positions, antennas_positions, 2 * P.M * sizeof(int), hipMemcpyHostToDevice);

        //Call GPU
        solutionEval<<<P.N / 1000, 1000>>>(dev_P, dev_antennas_positions, dev_building_score, dev_building_antenna);

        //Copy results back
        hipMemcpy(building_score, dev_building_score, P.N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(building_antenna, dev_building_antenna, P.N * sizeof(int), hipMemcpyDeviceToHost);

        long int score = 0;
        bool reward = 1;
        for (size_t b = 0; b < P.N; b++)
        {
            score += building_score[b];
            reward &= (building_antenna[b] != -1);
        }
        if (reward)
        {
            score += P.R;
        }

        if (score > best_score)
        {
            best_score = score;
            memcpy(best_antennas_positions, antennas_positions, 2 * P.M * sizeof(int));
            printf("Solution score: %ld\n", score);
            printf("Reward: %d\n", reward);
            printf("Relative distance %.4f\n", score * 1.0 / 2078043619);
        }
    }

    hipFree(dev_P.buildings_c);
    hipFree(dev_P.buildings_r);
    hipFree(dev_P.buildings_latency);
    hipFree(dev_P.buildings_connection_speed);
    hipFree(dev_P.antennas_range);
    hipFree(dev_P.antennas_speed);

    free(P.buildings_c);
    free(P.buildings_r);
    free(P.buildings_latency);
    free(P.buildings_connection_speed);
    free(P.antennas_range);
    free(P.antennas_speed);

    hipHostFree(antennas_positions);
    hipHostFree(building_score);
    hipHostFree(building_antenna);

    return 0;
}
